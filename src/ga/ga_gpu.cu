#include "hip/hip_runtime.h"
#include "ga_gpu.h"

#include "fitness_function.cuh"

#include "utility/cuda_core.cuh"
#include "utility/timer.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>

#include <algorithm>
#include <execution>

// Seeds the states. via
// https://docs.nvidia.com/cuda/hiprand/device-api-overview.html#device-api-example
__global__ static void initStates(const unsigned populationSize, hiprandState *state) {
	const auto tId = blockIdx.x * blockDim.x + threadIdx.x;

	/* Each thread gets same seed, a different sequence
	   number, no offset */
	if (tId < populationSize)
		hiprand_init(1234, tId, 0,
					&state[tId]); // NOLINT: 1234 is a very magical number indeed!
}

__device__ static Genome generateGenome(hiprandState *state) {
	Genome genome{};

	for (auto &gene : genome.genes)
		gene = hiprand(state);

	return genome;
}
__global__ static void populate(Genome *population, const unsigned populationSize,
								hiprandState *state) {
	const auto tId = blockIdx.x * blockDim.x + threadIdx.x;

	if (tId < populationSize) {
		hiprandState *localState = &state[tId];
		population[tId] = generateGenome(localState);
		state[tId] = *localState;
	}
}
__global__ static void calculateFitneess(Genome *population, const unsigned populationSize,
										 const unsigned eliteSize = 0) {
	const auto tId = blockIdx.x * blockDim.x + threadIdx.x;

	if (tId < populationSize - eliteSize) {
		auto genome = population[tId];
		genome.fitness = fitnessFunction(genome, tId);
		population[tId] = genome;
	}
}

template <typename T>
__device__ void warpSumReduce(volatile T *sharedData, unsigned tId) {
	sharedData[tId] += sharedData[tId + 32];
	sharedData[tId] += sharedData[tId + 16];
	sharedData[tId] += sharedData[tId + 8];
	sharedData[tId] += sharedData[tId + 4];
	sharedData[tId] += sharedData[tId + 2];
	sharedData[tId] += sharedData[tId + 1];
}
template <typename T>
__device__ void sumReduce(volatile T *sharedData, unsigned blocDim, unsigned tId) {
	for (auto s = blocDim / 2; s > GAGPU::warpSize; s >>= 1U) {
		if (tId < s)
			sharedData[tId] += sharedData[tId + s];
		__syncthreads();
	}

	if (tId < GAGPU::warpSize)
		warpSumReduce(sharedData, tId);
}
__global__ static void fitnessReduceStep1(Genome *population, const unsigned populationSize,
										  Genome::fitness_t *fitnessCumulative) {
	__shared__ Genome::fitness_t sharedSum[GAGPU::blockSize];
	const auto tId = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	if (tId < populationSize)
		sharedSum[threadIdx.x] = population[tId].fitness;
	else
		sharedSum[threadIdx.x] = 0;

	if (tId + blockDim.x < populationSize)
		sharedSum[threadIdx.x] += population[tId + blockDim.x].fitness;

	__syncthreads();

	sumReduce(sharedSum, blockDim.x, threadIdx.x);

	if (threadIdx.x == 0)
		fitnessCumulative[blockIdx.x] = sharedSum[0];
}
__global__ static void fitnessReduceStep2(Genome::fitness_t *fitnessCumulative,
										  const unsigned fitnessCumulativeSize) {
	__shared__ Genome::fitness_t sharedSum[GAGPU::blockSize];
	const auto tId = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	if (tId < fitnessCumulativeSize)
		sharedSum[threadIdx.x] = fitnessCumulative[tId];
	else
		sharedSum[threadIdx.x] = 0;

	if (tId + blockDim.x < fitnessCumulativeSize)
		sharedSum[threadIdx.x] += fitnessCumulative[tId + blockDim.x];

	__syncthreads();

	sumReduce(sharedSum, blockDim.x, threadIdx.x);

	if (threadIdx.x == 0)
		fitnessCumulative[blockIdx.x] = sharedSum[0];
}

__device__ static Genome rouletteSelect(Genome *population, const Genome::fitness_t cumulative,
										hiprandState *state) {
	const Genome::fitness_t selectionLocation = hiprand_uniform(state) * cumulative;
	Genome::fitness_t selectionIndex = 0;

	for (unsigned i = 0;; i++) {
		selectionIndex += population[i].fitness;
		if (selectionIndex >= selectionLocation)
			return population[i];
	}
}

__device__ static void crossOver(Genome &genomeA, Genome &genomeB, const unsigned crossPoint) {
	const Genome::dna_t tailMask = ~(Genome::dnaMax << crossPoint);

	for (unsigned i = 0; i < Genome::numberOfGenes; i++) {
		const Genome::dna_t difference = genomeA.genes[i] ^ genomeB.genes[i];
		const Genome::dna_t geneChanger = difference & tailMask;
		genomeA.genes[i] ^= geneChanger;
		genomeB.genes[i] ^= geneChanger;
	}
}

__device__ static void mutate(Genome &genome, const float mutationChance, hiprandState *state) {
	for (auto &gene : genome.genes)
		for (unsigned i = 0; i < numberOfBitsIn<Genome::dna_t>; i++)
			if (hiprand_uniform(state) <= mutationChance)
				gene ^= (Genome::dna_t(1) << i);
}

__global__ static void reproduce(Genome *population, const unsigned populationSize,
								 const unsigned eliteSize, const float mutationChance,
								 Genome::fitness_t *fitnessCumulative,
								 Genome::fitness_t greatestFitness, hiprandState *state) {
	__shared__ Genome sharedParents[GAGPU::blockSize];
	const auto tId = blockIdx.x * blockDim.x + threadIdx.x;
	const auto totalFitness =
		Genome::fitness_t(populationSize) - (fitnessCumulative[0] / greatestFitness);

	if (tId == populationSize - 1) {
		printf("Mean fitness: %.8f\n", fitnessCumulative[0] / Genome::fitness_t(populationSize));
		printf("Best fitness: %.8f\n", population[populationSize - 1].fitness);
		printf("Wrost fitness: %.8f\n", greatestFitness);
	}

	// fitness to fitness
	Genome::fitness_t fitness;
	if (tId < populationSize) {
		fitness = population[tId].fitness;
		population[tId].fitness = 1.0 - (fitness / greatestFitness);
	}

	if (tId < populationSize - eliteSize) {
		hiprandState *localState = &state[tId];

		sharedParents[threadIdx.x] = rouletteSelect(population, totalFitness, localState);

		__syncthreads();

		if (threadIdx.x % 2 == 0) {
			const unsigned crossOverMidPoint =
				hiprand_uniform(localState) * (numberOfBitsIn<Genome::dna_t> - 2UI64) + 1UI64;

			crossOver(sharedParents[threadIdx.x], sharedParents[threadIdx.x + 1],
					  crossOverMidPoint);
		}

		__syncthreads();

		mutate(sharedParents[threadIdx.x], mutationChance, localState);

		population[tId] = sharedParents[threadIdx.x];
		state[tId] = *localState;
	}

	// fitness to fitness (only elite necessary)
	if (tId < populationSize && tId >= populationSize - eliteSize)
		population[tId].fitness = fitness;
}

GAGPU::GAGPU(const unsigned populationSize, const unsigned eliteSize,
			 const float mutationChancePerGene)
	: populationSize_(populationSize), eliteSize_(eliteSize),
	  mutationChance_(mutationChancePerGene / numberOfBitsIn<decltype(Genome::genes)>),
	  populationHost_(populationSize) {

	cudaCall(hipMalloc(&populationDev_, sizeof(Genome) * populationSize_));
	cudaCall(hipMalloc(&fitnessCumulative_, sizeof(Genome::fitness_t) * fitnessCumulativeSize_));
	cudaCall(hipMalloc(&stateDev_, sizeof(hiprandState) * populationSize_));

	initStates<<<gridSize_, GAGPU::blockSize>>>(populationSize_, stateDev_);
	afterKernelCall();

	generatePopulation();
}

void GAGPU::generatePopulation() {
	populate<<<gridSize_, GAGPU::blockSize>>>(populationDev_, populationSize_, stateDev_);
	afterKernelCall();

	calculateFitneess<<<gridSize_, GAGPU::blockSize>>>(populationDev_, populationSize_);
	afterKernelCall();
}

std::vector<Genome> GAGPU::getPopulation() {
	cudaCall(hipMemcpy(populationHost_.data(), populationDev_, sizeof(Genome) * populationSize_,
						hipMemcpyDeviceToHost));

	return populationHost_;
}

GAGPU::~GAGPU() {
	cudaCall(hipFree(populationDev_));
	cudaCall(hipFree(fitnessCumulative_));
	cudaCall(hipFree(stateDev_));
}

void GAGPU::run(const unsigned numberOfGenerations) {
	for (unsigned generation = 0; generation < numberOfGenerations; generation++) {
		Timer t{"generation loop"};
		printf("\ngeneration: %u\n", generation);

		const auto greatestFitness = getGreatestFitnessAndSelectElite();

		fitnessReduceStep1<<<fitnessCumulativeSize_, GAGPU::blockSize>>>(
			populationDev_, populationSize_, fitnessCumulative_);
		afterKernelCall();
		fitnessReduceStep2<<<1, GAGPU::blockSize>>>(fitnessCumulative_, fitnessCumulativeSize_);
		afterKernelCall();

		// TEST
		//{
		//	hipDeviceSynchronize();

		//	getPopulation();

		//	auto totalFitness = std::transform_reduce(
		//		populationHost_.begin(), populationHost_.end(),
		// Genome::fitness_t{0}, std::plus{},
		//		[](const Genome &genome) { return genome.fitness; });
		//	printf("total fitness CPU: %.8f\n", totalFitness);

		//}

		reproduce<<<gridSizeNoElite_, GAGPU::blockSize>>>(
			populationDev_, populationSize_, eliteSize_, mutationChance_, fitnessCumulative_,
			greatestFitness, stateDev_);
		afterKernelCall();

		hipDeviceSynchronize();

		calculateFitneess<<<gridSizeNoElite_, GAGPU::blockSize>>>(populationDev_, populationSize_,
																  eliteSize_);
		afterKernelCall();
	}
}

Genome::fitness_t GAGPU::getGreatestFitnessAndSelectElite() {
	cudaCall(hipMemcpy(populationHost_.data(), populationDev_, sizeof(Genome) * populationSize_,
						hipMemcpyDeviceToHost));

	std::partial_sort_copy(std::execution::par_unseq, populationHost_.rbegin(),
						   populationHost_.rend(), populationHost_.rbegin(),
						   populationHost_.rbegin() + eliteSize_);

	cudaCall(hipMemcpy(populationDev_, populationHost_.data(), sizeof(Genome) * populationSize_,
						hipMemcpyHostToDevice));

	const Genome &wrost =
		*std::max_element(std::execution::par_unseq, populationHost_.begin(), populationHost_.end(),
						  [](const Genome &a, const Genome &b) { return a.fitness < b.fitness; });

	return wrost.fitness;
}